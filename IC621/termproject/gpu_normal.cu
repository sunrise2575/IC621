// Streams
#include <iostream>
#include <sstream>
#include <fstream>

// Containers
#include <string>
#include <vector>

// Time
#include <chrono>

// C headers
#include <cmath>
#include <cstdlib>
#include <cstring>

// CUDA headers
#include <hip/hip_runtime.h>



int reverse_int32(const int i) {
	unsigned char byte1, byte2, byte3, byte4;
	byte1 = i & 255;
	byte2 = (i >> 8) & 255;
	byte3 = (i >> 16) & 255;
	byte4 = (i >> 24) & 255;
	return ( (int)byte1 << 24 ) + ( (int)byte2 << 16 ) + ( (int)byte3 << 8 ) + (int)byte4;
}

struct image_t {
	int magic_number;
	int size, row, column;
	float** data;
	image_t(const char* file_path, const int padding_width) {
		std::ifstream infile(file_path, std::ios::binary);
		if (!infile.is_open()) {
			printf("----ERROR: File open failure\n");
			exit(1);
		}

		infile.read((char*)&this->magic_number, sizeof(this->magic_number));
		this->magic_number = reverse_int32(this->magic_number);

		infile.read((char*)&this->size, sizeof(this->size));
		this->size = reverse_int32(this->size);

		infile.read((char*)&this->row, sizeof(this->row));
		this->row = reverse_int32(this->row);

		infile.read((char*)&this->column, sizeof(this->column));
		this->column = reverse_int32(this->column);

		int new_row = this->row + 2 * padding_width;
		int new_column = this->column + 2 * padding_width;

		this->data = (float**)malloc(sizeof(float*) * this->size);
		for(int i = 0; i < this->size; i++) {
			this->data[i] = (float*)malloc(sizeof(float) * new_row * new_column); // 패딩 할 걸 생각해서 잡는다
			memset(this->data[i], 0, sizeof(float) * new_row * new_column);
		}

		for(int image = 0; image < this->size; ++image) {
			for(int i = 0; i < this->row; ++i) {
				for(int j = 0; j < this->column; ++j) {
					unsigned char temp = 0;
					infile.read((char*)&temp, sizeof(temp));
					this->data[image][(i + padding_width) * new_row + (j + padding_width)] = (float)temp * (float)((float)1/(float)255);
				}
			}
		}

		this->row = new_row;
		this->column = new_column;

		infile.close();
	}

	~image_t() {
		for(int i = 0; i < size; i++) {
			free(this->data[i]);
		}
		free(this->data);
	}
};

struct label_t {
	int magic_number;
	int size;
	int* data;
	label_t(const char* file_path) {
		std::ifstream infile(file_path, std::ios::binary);
		if (!infile.is_open()) {
			printf("----ERROR: File open failure\n");
			exit(1);
		}

		infile.read((char*)&this->magic_number, sizeof(this->magic_number));
		this->magic_number = reverse_int32(this->magic_number);

		infile.read((char*)&this->size, sizeof(this->size));
		this->size = reverse_int32(this->size);

		this->data = (int*)malloc(sizeof(int) * this->size);

		for(int i = 0; i < this->size; ++i) {
			unsigned char temp = 0;
			infile.read((char*)&temp, sizeof(temp));
			this->data[i] = (int)temp;
		}

		infile.close();
	}
	~label_t() {
		if (this->data != nullptr) {
			free(this->data);
		}
	}
};

struct result_layers_t {
	float input[1 * 32 * 32];	// 입력 이미지 (제로패딩 된 것)
	float conv_1[6 * 28 * 28];	// 콘볼루션 결과 1
	float sub_1[6 * 14 * 14];	// 서브샘플링 결과 1
	float conv_2[16 * 10 * 10];	// 콘볼루션 결과 2
	float sub_2[16 * 5 * 5];	// 서브샘플링 결과 2
	float full_1[120];			// 뉴런 1
	float full_2[84];			// 뉴런 2
	float output[10];			// 출력
};

struct mask_bias_weight_t {
	float mask_1[6 * 1 * 5 * 5];	// 콘볼루션 마스크 1
	float bias_1[6];				// 바이어스 1
	float mask_2[16 * 6 * 5 * 5];	// 콘볼루션 마스크 2
	float bias_2[16];				// 바이어스 2
	float weight_1[120 * 400];		// 웨이트 1
	float bias_3[120];				// 바이어스 3
	float weight_2[84 * 120];		// 웨이트 2
	float bias_4[84];				// 바이어스 4
	float weight_3[10 * 84];		// 웨이트 3
	float bias_5[10];				// 바이어스 5
};

void print_mnist(const float* data, const int label) {
	std::cout << "Check data for label " << label << std::endl;
	for(int r = 0; r < 32; r++) {
		for (int c = 0; c < 32; c++) {
			if (data[r * 32 + c] > 0.5f) {
				std::cout << "■";
			} else {
				std::cout << "□";
			}
		}
		std::cout << std::endl;
	}
}

void parse_mask_bias_weight(const char* file_path, mask_bias_weight_t* filter) {
	std::ifstream infile(file_path);
	if (!infile.is_open()) {
		printf("----ERROR: File open failure\n");
		exit(1);
	}

	float* temp = (float*)malloc(sizeof(mask_bias_weight_t));

	std::string each_line;
	double num;
	int word_count = 0;
	while (std::getline(infile, each_line)) {
		if (each_line.length() == 0) { continue; }
		if (each_line[0] == '#') { continue; }

		std::stringstream ss;
		ss.str(each_line);
		while (ss >> num) {
			temp[word_count] = (float)num;
			word_count++;
		}
	}

	memcpy(filter, temp, sizeof(mask_bias_weight_t));

	free(temp);

	infile.close();
}

__device__ float sigmoid(float x) {
	return 1.0 / (1.0 + exp(-x));
}

__global__ void conv_forward(float* input, int input_count, int input_row, int input_column,
							 float* kernel, int kernel_count1, int kernel_count2, int kernel_row, int kernel_column,
							 float* output, int output_count, int output_row, int output_column)
{
	int m = blockIdx.x; // output image number
	int h = threadIdx.x; // output height
	int w = threadIdx.y; // output width

	float accumulated = 0;
	for (int c = 0; c < input_count; c++) {
		for (int p = 0; p < kernel_row; p++) {
			for (int q = 0; q < kernel_column; q++) {
				accumulated += input[input_row * input_column * c + input_column * (h + p) + (w + q)]
				* kernel[kernel_count2 * kernel_row * kernel_column * m + kernel_row * kernel_column * c + kernel_column * p + q];
			}
		}
	}
	output[output_row * output_column * m + output_column * h + w] = accumulated;

}

__global__ void pool_forward(float* input, int input_count, int input_row, int input_column,
							 float* bias,
							 float* output, int output_count, int output_row, int output_column)
{
	int m = blockIdx.x; // output image number
	int h = threadIdx.x; // output height
	int w = threadIdx.y; // output width

	float accumulated = 0;
	for (int p = 0; p < 2; p++) {
		for (int q = 0; q < 2; q++) {
			accumulated += input[input_row * input_column * m + input_column * (2 * h + p) + (2 * w + q)] / (2 * 2);
		}
	}
	output[output_row * output_column * m + output_column * h + w]
	= sigmoid(accumulated + bias[m]);
}

__device__ void full_forward(float* input,
							 float* weight, int weight_row, int weight_column,
							 float* bias,
							 float* output)
{
	int i = threadIdx.x; // weight row
	if (i < weight_row) {
		for (int k = 0; k < weight_column; k++) {
			output[i] += weight[weight_column * i + k] * input[k];
		}
		output[i] = sigmoid(output[i] + bias[i]);
	}
}

__global__ void full_forward_total(result_layers_t* d_result, mask_bias_weight_t* d_filter) {
	int image = blockIdx.x;

	// 서브샘플링과 웨이트 및 바이어스로 다음 뉴런 연산
	full_forward(d_result[image].sub_2,
				 d_filter->weight_1, 120, 400,
				 d_filter->bias_3,
				 d_result[image].full_1);

	__syncthreads();

	// 뉴런 결과와 웨이트 및 바이어스로 다음 뉴런 연산
	full_forward(d_result[image].full_1,
				 d_filter->weight_2, 84, 120,
				 d_filter->bias_4,
				 d_result[image].full_2);

	__syncthreads();


	// 뉴런 결과와 웨이트 및 바이어스로 출력 연산
	full_forward(d_result[image].full_2,
				 d_filter->weight_3, 10, 84,
				 d_filter->bias_5,
				 d_result[image].output);

	__syncthreads();
}

int find_max_index(float* input, const int size) {
	int result = 0;
	float max_value = -9.99;
	for (int i = 0; i < size; i++) {
		if (input[i] > max_value) {
			max_value = input[i];
			result = i;
		}
	}

	return result;
}

int main() {
	// MNIST 파일 불러오기
    image_t train_image("/home/ic621/mnist/train-images-idx3-ubyte", 2);
    label_t train_label("/home/ic621/mnist/train-labels-idx1-ubyte");
    image_t test_image("/home/ic621/mnist/t10k-images-idx3-ubyte", 2);
	label_t test_label("/home/ic621/mnist/t10k-labels-idx1-ubyte");

    // result_layers에 적재하기
    result_layers_t* train_result = (result_layers_t*)malloc(sizeof(result_layers_t) * train_image.size);
    result_layers_t* test_result = (result_layers_t*)malloc(sizeof(result_layers_t) * test_image.size);

    for (int image = 0; image < train_image.size; image++) {
        for (int i = 0; i < train_image.row; i++) {
            for (int j = 0; j < train_image.column; j++) {
                train_result[image].input[32 * i + j] = train_image.data[image][i * train_image.row + j];
            }
        }
    }

    for (int image = 0; image < test_image.size; image++) {
        for (int i = 0; i < test_image.row; i++) {
            for (int j = 0; j < test_image.column; j++) {
                test_result[image].input[32 * i + j] = test_image.data[image][i * test_image.row + j];
            }
        }
    }

	// 필터 불러오기
	mask_bias_weight_t filter;
	parse_mask_bias_weight("text_out_lenet5.txt", &filter);

	int total = test_image.size;

	// test_result와 filter모두 cuda로 보내기
	result_layers_t* d_result;
	mask_bias_weight_t* d_filter;

	hipMalloc(&d_result, sizeof(result_layers_t) * total);
	hipMalloc(&d_filter, sizeof(mask_bias_weight_t));

	hipMemcpy(d_result, test_result, sizeof(result_layers_t) * total, hipMemcpyHostToDevice);
	hipMemcpy(d_filter, &filter, sizeof(mask_bias_weight_t), hipMemcpyHostToDevice);

	auto start = std::chrono::high_resolution_clock::now();

	for (int image = 0; image < total; image++) {
		// 입력이미지와 콘볼루션 마스크로 콘볼루션 연산 수행
		{
			dim3 blocks(6, 1, 1); // output map 개수를 써준다
			dim3 threads(28, 28, 1); // output row/column 개수를 써준다
			conv_forward<<<blocks, threads>>>(d_result[image].input, 1, 32, 32,
											  d_filter->mask_1, 6, 1, 5, 5,
											  d_result[image].conv_1, 6, 28, 28);
		}

		// 콘볼루션 결과와 바이어스로 서브샘플링 수행
		{
			dim3 blocks(6, 1, 1); // output map 개수를 써준다
			dim3 threads(14, 14, 1); // output row/column 개수를 써준다
			pool_forward<<<blocks, threads>>>(d_result[image].conv_1, 6, 28, 28,
				d_filter->bias_1,
				d_result[image].sub_1, 6, 14, 14);
		}

		// 서브샘플링 결과와 콘볼루션 마스크로 콘볼루션 연산 수행
		{
			dim3 blocks(16, 1, 1); // output map 개수를 써준다
			dim3 threads(10, 10, 1); // output row/column 개수를 써준다
			conv_forward<<<blocks, threads>>>(d_result[image].sub_1, 6, 14, 14,
											  d_filter->mask_2, 16, 6, 5, 5,
											  d_result[image].conv_2, 16, 10, 10);
		}

		// 콘볼루션 결과와 바이어스로 서브샘플링 수행
		{
			dim3 blocks(16, 1, 1); // output map 개수를 써준다
			dim3 threads(5, 5, 1); // output row/column 개수를 써준다
			pool_forward<<<blocks, threads>>>(d_result[image].conv_2, 16, 10, 10,
				d_filter->bias_2,
				d_result[image].sub_2, 16, 5, 5);
		}
	}

	// fully connected는 갈수록 행렬의 크기가 감소하며, 계산량이 많지 않기에,
	// image별로 block을 생성하여 한꺼번에 계산하였다.
	// block개수는 65536이 최대. test가 10000개 이미지이므로 괜찮다.
	{
		dim3 blocks(total, 1, 1);
		dim3 threads(120, 1, 1);
		full_forward_total<<<blocks, threads>>>(d_result, d_filter);
	}

	auto end = std::chrono::high_resolution_clock::now();
	auto elapsed = std::chrono::duration_cast<std::chrono::duration<double> >(end - start);

	// GPU에서 계산한 거를 옮긴다
	hipMemcpy(test_result, d_result, sizeof(result_layers_t) * total, hipMemcpyDeviceToHost);

	// 정확도를 구한다
	int accurate = 0;
	for (int image = 0; image < total; image++) {
		int result = find_max_index(test_result[image].output, 10);
		int answer = test_label.data[image];
		if (result == answer) {
			accurate++;
		}
	}

	printf("test data accuracy: %8lf\n", (float)accurate / (float)total);
	printf("elapsed time      : %8lf (sec)\n", elapsed.count());

    free(train_result);
    free(test_result);

    return 0;
}
