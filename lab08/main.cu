#include <hip/hip_runtime.h>



#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <iostream>
#include <chrono>

#include <cstdlib>
#include <cmath>

// the max number of (x,y) threads is 1024
// which is 1024 = 32 x 32, so 0 <= threadIdx.x < 32 and 0 <= threadIdx.y < 32
constexpr unsigned int THREAD_COUNT = 32;

#define TILE_WIDTH 32

__global__ void kernel_fill_random(float* matrix, const unsigned int width) {
	// index
	const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;

	// set random seed
	unsigned int seed = i + j;
	hiprandState s;

	hiprand_init(seed, 0, 0, &s);

	//fill matrix
	if (i < width && j < width) {
		matrix[i * width + j] = hiprand_uniform(&s);
	}
}

__global__ void kernel_naive(const float* M, const float* N, float* P, const unsigned int width) {
	// index
	const unsigned int i = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < width && j < width) {
		float result = 0;

		for (int k = 0; k < width; ++k) {
			result += M[i * width + k] * N[k * width + j];
		}

		P[i * width + j] = result;
	}

}

__global__ void kernel_tiled(const float* M, const float* N, float* P, const int width) {
	__shared__ float Ms[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Ns[TILE_WIDTH][TILE_WIDTH];

	int bx = blockIdx.x; int by = blockIdx.y;
	int tx = threadIdx.x; int ty = threadIdx.y;

	int row = by * TILE_WIDTH + ty;
	int col = bx * TILE_WIDTH + tx;

	float result = 0;
	const unsigned int TILE_COUNT = (unsigned int)std::ceil((float)width/(float)TILE_WIDTH);
	for (int m = 0; m < TILE_COUNT; ++m) {
		if (m * TILE_WIDTH + tx < width && row < width) {
			Ms[ty][tx] = M[row * width + (m * TILE_WIDTH + tx)];
		} else {
			Ms[ty][tx] = 0.0;
		}

		if (m * TILE_WIDTH + ty < width && col < width) {
			Ns[ty][tx] = N[col + (m * TILE_WIDTH + ty) * width];
		} else {
			Ns[ty][tx] = 0.0;
		}
		__syncthreads();

		for (int k = 0; k < TILE_WIDTH; ++k) {
			result += Ms[ty][k] * Ns[k][tx];
		}
		__syncthreads();
	}

	if (row < width && col < width) {
		P[row * width + col] = result;
	}
}

double naive(const float* d_M, const float* d_N, float* d_P, const unsigned int width) {
	const unsigned int block_count = (unsigned int)std::ceil((float)width/(float)THREAD_COUNT);
	dim3 blocks(block_count, block_count);
	dim3 threads(THREAD_COUNT, THREAD_COUNT);

	auto start = std::chrono::high_resolution_clock::now();

	kernel_naive<<<blocks, threads>>>(d_M, d_N, d_P, width);
	hipDeviceSynchronize();

	auto end = std::chrono::high_resolution_clock::now();
	auto elapsed = std::chrono::duration_cast<std::chrono::duration<double> >(end - start);

	return elapsed.count();
}

double tiled(const float* d_M, const float* d_N, float* d_P, const unsigned int width) {
	const unsigned int block_count = (unsigned int)std::ceil((float)width/(float)TILE_WIDTH);
	dim3 blocks(block_count, block_count);
	dim3 threads(TILE_WIDTH, TILE_WIDTH);

	auto start = std::chrono::high_resolution_clock::now();

	kernel_tiled<<<blocks, threads>>>(d_M, d_N, d_P, width);
	hipDeviceSynchronize();

	auto end = std::chrono::high_resolution_clock::now();
	auto elapsed = std::chrono::duration_cast<std::chrono::duration<double> >(end - start);

	return elapsed.count();
}

void file_load(const char* filepath, float* matrix, const size_t size) {
	FILE* file;
	size_t result;

	file = fopen(filepath, "rb");
	if (!file) {
		printf(">>>ERROR: opening %s\n", filepath);
		exit(1);
	}

	fseek(file, 0, SEEK_END);
	result = ftell(file);
	rewind(file);

	if (size != result) {
		printf(">>>ERROR: wrong filesize %ld\n", size);
		exit(1);
	}

	result = fread(matrix, 1, size, file);
	if (result != size) {
		printf(">>>ERROR: reading %s\n", filepath);
		exit(1);
	}

	fclose(file);
}

void file_save(float* matrix, const char* filepath, const size_t size) {
	FILE* file;
	size_t result;

	file = fopen(filepath, "wb");
	if (!file) {
		printf(">>>ERROR: opening %s\n", filepath);
		exit(1);
	}

	result = fwrite(matrix, 1, size, file);
	if (result != size) {
		printf(">>>ERROR: writing %s\n", filepath);
		exit(1);
	}

	fclose(file);
}

bool is_same(const float* matrix1, const float* matrix2, const size_t width) {
	for (size_t i = 0; i < width * width; i++) {
		if (matrix1[i] != matrix2[i]) {
			printf("position: %ld\n", i);
			return false;
		}
	}

	return true;
}

int main(int argc, char** argv) {
	if (argc != 5) { return 0; }

	const char* filepath_M = argv[1];
	const char* filepath_N = argv[2];
	const char* filepath_P = argv[3];
	const unsigned int width = atoi(argv[4]);

	const size_t size = width * width * sizeof(float);

	float* M = (float*)malloc(size);
	float* N = (float*)malloc(size);
	float* P1 = (float*)malloc(size);
	float* P2 = (float*)malloc(size);
	printf(">  COMPLETE: malloc on host\n");

	file_load(filepath_M, M, size);
	printf(">  COMPLETE: load %s\n", filepath_M);

	file_load(filepath_N, N, size);
	printf(">  COMPLETE: load %s\n", filepath_N);

	float *d_M, *d_N, *d_P;
	hipMalloc(&d_M, size); hipMalloc(&d_N, size); hipMalloc(&d_P, size);
	hipDeviceSynchronize();
	printf(">  COMPLETE: malloc on device\n");

	hipMemcpy(d_M, M, size, hipMemcpyHostToDevice);
	hipMemcpy(d_N, N, size, hipMemcpyHostToDevice);
	printf(">  COMPLETE: memcpy to device\n");

	double elapsed_naive = naive(d_M, d_N, d_P, width);
	printf(">>>RESULT: naive elapsed time: %8lf (sec)\n", elapsed_naive);
	hipMemcpy(P1, d_P, size, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	double elapsed_tiled = tiled(d_M, d_N, d_P, width);
	printf(">>>RESULT: elapsed time: %8lf (sec)\n", elapsed_tiled);
	hipMemcpy(P2, d_P, size, hipMemcpyDeviceToHost);

	if (is_same(P1, P2, width)) {
		printf(">  COMPLETE: no difference between results of naive and tiled method\n");
	} else {
		printf(">>>ERROR: difference between results of naive and tiled method\n");
		exit(1);
	}

	file_save(P2, filepath_P, size);
	printf(">  COMPLETE: save result matrix on %s\n", filepath_P);

	// free memory on device
	hipFree(d_M); hipFree(d_N); hipFree(d_P);

	// free memory on host
	free(M); free(N); free(P1); free(P2);
}
